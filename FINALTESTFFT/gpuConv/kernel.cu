#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <cmath>


using namespace std;

//defines for test data 2
#define samp_rate 8000
#define bits_per_sample 16
#define ch 1
#define avg_bytes_per_sec 16000



/* ---- ---- AUDIO FILE OPERATIONS -- ---- ---- */

// Function to read a WAV file and extract audio data
vector<float> readWavFile(const string& filename) {
    ifstream file(filename, ios::binary);
    if (!file) {
        cerr << "Error opening file: " << filename << endl;
        return {};
    }

    // Check if the file starts with the RIFF header
    char header[4];
    file.read(header, 4);
    if (string(header, 4) != "RIFF") {
        cerr << "Not a valid WAV file." << endl;
        return {};
    }

    // Seek to the start of the data chunk (skipping other headers)
    file.seekg(40, ios::beg);

    // Read the audio data
    vector<float> audioData;
    int16_t sample;
    while (file.read(reinterpret_cast<char*>(&sample), sizeof(sample))) {
        // Convert the 16-bit signed integer sample to 16-bit signed floating-point
        float floatSample = static_cast<float>(sample) / 32768.0f; // Normalization
        audioData.push_back(floatSample);
    }

    file.close();
    return audioData;
}

//write intgers in spcific amount of bytes
void writeToFile(ofstream& file, int value, int size)
{
    file.write(reinterpret_cast<const char*> (&value), size);
}

//write audio data to a file
void writeWavFile(vector<float> audioData, string filename)
{
    // You can now use audioData as an array of signed 16-bit integers
    ofstream audioFile;
    audioFile.open(filename, ios::binary);

    //header chunk
    audioFile << "RIFF"; // standard ID of the header chunk (4 bytes)
    audioFile << "----"; // size of the file (unknown yet)
    audioFile << "WAVE"; //data of the header chunck

    //format chunk
    audioFile << "fmt "; // standard ID of the format chunk (4 bytes with space)
    writeToFile(audioFile, 16, 4); // standard size of format chunk is 16 bytes
    writeToFile(audioFile, 1, 2); // compression code 1 = geen compressie
    writeToFile(audioFile, ch, 2); // channels
    writeToFile(audioFile, samp_rate, 4); // sample rate
    writeToFile(audioFile, avg_bytes_per_sec, 4); // avg_bytes_per_sec
    writeToFile(audioFile, 2, 2); // block align
    writeToFile(audioFile, bits_per_sample, 2); //bit depth

    //data chunck
    audioFile << "data";
    audioFile << "----";

    int prepos = audioFile.tellp(); // get current position in file

    //write the collected audio data to the new file
    for (int i = 0; i < audioData.size(); i++) {
        int16_t sample = static_cast<int16_t>(audioData[i] * 32768.0f);
        audioFile.write(reinterpret_cast<const char*>(&sample), sizeof(sample));
    }


    int postpos = audioFile.tellp(); // get current position in file

    audioFile.seekp(prepos - 4); // jump to 4 bytes before prepos and overwrite it with size)
    writeToFile(audioFile, postpos - prepos, 4);

    audioFile.seekp(4, ios::beg); // jump to the beginning of the file, offsetted with 4 bytes
    writeToFile(audioFile, postpos - 8, 4); // postpos is last position (so entire size), substract the 8 bytes of ID and size

    audioFile.close();
}

/* ---- ---- END AUDIO FILE OPERATIONS---- ---- */



/* ---- ---- GPU KERNEL + OPERATIONS- ---- ---- */

// naive convolution kernel (without shared/constant memory, or tiling)
__global__ void convolutionKernel(const float* signal, const float* impulse_response, float* output, int signalSize, int impulseSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < signalSize + impulseSize - 1) {
        for (int j = 0; j < impulseSize; ++j) {
            if (idx - j >= 0 && idx - j < signalSize) {
                atomicAdd(&output[idx], signal[idx - j] * impulse_response[j]);
            }
        }
    }
}

//function that handles the gpu memcopys, mallocs and calls
vector<float> convolution(const std::vector<float>& signal, const std::vector<float>& impulse_response) {
    std::vector<float> output(signal.size() + impulse_response.size() - 1, 0.0f);

    float* d_signal, * d_impulse_response, * d_output;

    hipMalloc((void**)&d_signal, signal.size() * sizeof(float));
    hipMalloc((void**)&d_impulse_response, impulse_response.size() * sizeof(float));
    hipMalloc((void**)&d_output, output.size() * sizeof(float));

    hipMemcpy(d_signal, signal.data(), signal.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_impulse_response, impulse_response.data(), impulse_response.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output, output.data(), output.size() * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (signal.size() + impulse_response.size() - 1 + blockSize - 1) / blockSize;

    convolutionKernel << <numBlocks, blockSize >> > (d_signal, d_impulse_response, d_output, signal.size(), impulse_response.size());

    hipMemcpy(output.data(), d_output, output.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_signal);
    hipFree(d_impulse_response);
    hipFree(d_output);

    return output;
}

/* ---- ---- END GPU KERNEL + OPERATIONS-- ---- */



/* ---- ---- NORMALIZATION- ---- ---- ---- ---- */

// Function to calculate L2 norm of a vector
float calculateL2Norm(const std::vector<float>& vec) {
    float sum = 0.0;
    for (const auto& value : vec) {
        sum += value * value;
    }
    return sqrt(sum);
}

/* ---- ---- END NORMALIZATION-- ---- ---- ---- */



/* ---- ---- ---- ---- MAIN ---- ---- ---- ---- */

int main()
{
    // Provide the WAV file paths
    string filenameAudio = "AUDIO2.wav";
    string filenameImpulseResponse = "IR.wav";

    // Read the WAV files and extract audio data
    vector<float> audioData = readWavFile(filenameAudio);
    vector<float> irData = readWavFile(filenameImpulseResponse);

    // Check if audio data is extracted successfully
    if (audioData.empty() || irData.empty()) {
        cerr << "Failed to read audio data from files." << endl;
        return 1;
    }

    float irL2Norm = calculateL2Norm(irData);
    vector<float> normalizedIr;
    for (const auto& sample : irData) {
        normalizedIr.push_back(sample / irL2Norm);
    }

    // Print the number of audio samples extracted
    cout << "Number of audio samples in audio: " << audioData.size() << endl;
    cout << "Number of audio samples in impulse response: " << irData.size() << endl;

    // Perform convolution
    //vector<float> out = conv(audioData, normalizedIr);
    //vector<float> out = convolution(audioData, normalizedIr);
    //vector<float> out = conv(normalizedAudio, normalizedIr);

    // Perform convolution 10000 times and measure the time taken everytime, write the results to a file
    ofstream timeFile;
    timeFile.open("timeResults.txt");
    for (int i = 0; i < 100; i++) {
        cout << "Iteration " << i + 1 << endl;
        auto start = chrono::high_resolution_clock::now();
        vector<float> out = convolution(audioData, normalizedIr);
        auto end = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed = end - start;
        timeFile << elapsed.count() << endl;
    }

    // Write the output to a WAV file
    //writeWavFile(out, "OUTPUT_TEST_3.wav");

    return 0;
}

/* ---- ---- ---- ---- END MAIN- ---- ---- ---- */
